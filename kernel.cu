#include <hip/hip_runtime.h>

#include <hip/hip_runtime_api.h>

#include <hip/device_functions.h>
#pragma once
#ifdef __INTELLISENSE__
void __syncthreads();
long long int clock64();
#endif

#include <math.h>
#include <stdio.h>
#include <hiprand/hiprand_kernel.h>

// additional helping macros
#define MAX_BLOCKS 65535
#define HANDLE_ERROR(status) if(status != hipSuccess) printf("\n Error...")

//algorithm parameters i.e constants for scheme accuracy and parallel processing
#define THREADS 100
#define TRAJECTORIES 1000
#define n 100
#define N 1000

//parameters for SDE of the form 'dXt = a(t,Xt)*dt + b(t,Xt)dWt'
#define alpha 0.1209
#define mu 0.0423
#define sigma 0.1642 
#define T 1.0
#define x0 1.0

__device__ double func_a(double t, double x)
{
    return alpha*(mu - x);
}

__device__ double func_b(double t, double x)
{
    return sigma*sqrt(x);
}

//returns squared difference value between exact and approximated solutions
__device__ double scheme_error(double* exact, double* non_exact)
{
    double result = 0.0;
    int i;
    for (i = 0; i < n; i++)
    {
        result += pow(exact[i] - non_exact[i], 2);
    }
    return result;
}

//assigns vector of approximated realizations of XT to the solutions argument
__global__ void classic_euler(double* solutions)
{
    double X = x0;
    hiprandState_t state;
    hiprand_init(clock64() * blockDim.x, threadIdx.x, 0, &state);
    int j;
    double H = T / (n * N);
    for (j = 0; j < N; j++)
    {
        X += func_a(j*H, X) * H + func_b(j * H, X) * hiprand_normal(&state) * sqrt(H);
    }
    solutions[blockIdx.x] = X;
}

//prints given number of samples of the approximated solutions which are computed due to parallel processing
//additionally capable of saving it to the file if needed
void get_solution_samples(int samples, bool save_to_file=true, bool print=false)
{
    FILE* fptr = fopen("samples.txt", "w");;
    double* solutions = (double*) malloc(sizeof(double)*samples);
    double* solutions_dev;
    HANDLE_ERROR(hipMalloc((void**)&solutions_dev, sizeof(double) * samples));
    if (samples < MAX_BLOCKS) {
        classic_euler<<<samples, 1 >>>(solutions_dev);
        HANDLE_ERROR(hipMemcpy(solutions, solutions_dev, sizeof(double*) * samples, hipMemcpyDeviceToHost));
    }
    HANDLE_ERROR(hipFree(solutions_dev));
    int i;
    for (i = 0; i < samples; i++) {
        if(print) printf("%f\n", solutions[i]);
        if(save_to_file) fprintf(fptr, "%d,%f\n", i+1, solutions[i]);
    }
    free(solutions);
    fclose(fptr);
}

// partial_mean- mean value of the solutions for every block, partial_error - mean error of the solutions per every block
__global__ void euler_for_unknown(double* partial_mean, double* partial_error)
{
    __shared__ double error_for_thread[THREADS];
    __shared__ double solution_for_thread[THREADS];

    int index = threadIdx.x + blockDim.x * blockIdx.x;
    if (index < TRAJECTORIES)
    {
        hiprandState_t state;
        hiprand_init(clock64()*blockIdx.x, threadIdx.x, 0, &state);

        double temp = x0, TEMP = x0;
        double* x = (double*)malloc(sizeof(double) * n); // rozwi�zania na rzadkiej
        double* X = (double*)malloc(sizeof(double) * n); // rozwi�zania na g�stej

        double curr_W = 0.0;
        double prev_W = 0.0;
        double dW = 0.0;

        double h = T / n;
        double H = h / N;

        int i, j;
        for (i = 0; i < n; i++) {
            for (j = 0; j < N; j++)
            {
                dW = hiprand_normal(&state) * sqrt(H);
                TEMP += func_a(j * H, TEMP) * H + func_b(j * H, TEMP) * dW;
                curr_W += dW;
            }

            temp += func_a(i * h, temp) * h + func_b(i * h, temp) * (curr_W - prev_W);
            prev_W = curr_W;

            x[i] = temp;
            X[i] = TEMP;
        }

        solution_for_thread[threadIdx.x] = X[n - 1];
        error_for_thread[threadIdx.x] = scheme_error(X, x);
        free(x);
        free(X);
    }

    __syncthreads();

    if (threadIdx.x == 0)
    {
        //First thread is the one to manage reduction process
        double error_value = 0.0;
        double mean_value = 0.0;
        int i;
        for (i = 0; i < blockDim.x; i++)
        {
            if (i + blockDim.x * blockIdx.x < TRAJECTORIES) {
                error_value += error_for_thread[i] / blockDim.x;
                mean_value += solution_for_thread[i] / blockDim.x;
            }
        }
        partial_error[blockIdx.x] = error_value;
        partial_mean[blockIdx.x] = mean_value;
    }
}

//returns true if the device satisfies properties for parallel processing
bool is_enough_threads()
{
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);
    printf("Simulation running on a device: %s\n", prop.name);
    int max_threads_per_block = prop.maxThreadsPerBlock;
    printf("Available threads: %d\n=============\n\n", max_threads_per_block);
    return THREADS < max_threads_per_block;
}

// prints overall mean value of XT as well as approximation error
void print_results(double* partial_mean, double* partial_error, int size)
{
    int i;
    double mean = 0.0, error = 0.0;
    for (i = 0; i < size; i++)
    {
        error += partial_error[i] * THREADS / TRAJECTORIES;
        mean += partial_mean[i] * THREADS / TRAJECTORIES;
    }
    printf("error: %f\nmean: %f", error, mean);
}

int main()
{
    //Validating if accuracy parameters aren't too demanding
    if (!is_enough_threads()) {
        printf("Requesting for too many threads: %d", THREADS);
        return -1;
    }

    const int BLOCKS = (TRAJECTORIES + THREADS - 1) / THREADS;
    if (BLOCKS > MAX_BLOCKS) {
        printf("Requesting for too many blocks: %d", BLOCKS);
        return -1;
    }

    //Memory allocation
    double* partial_mean = (double*) malloc(sizeof(double*)*BLOCKS);
    double* partial_mean_dev;

    double* partial_error = (double*)malloc(sizeof(double*)*BLOCKS);
    double* partial_error_dev;

    //Events initialization for measuring time performance during kernel evaluation
    hipEvent_t start, stop;
    HANDLE_ERROR(hipEventCreate(&start));
    HANDLE_ERROR(hipEventCreate(&stop));

    HANDLE_ERROR(hipEventRecord(start, 0));

    HANDLE_ERROR(hipMalloc((void**)&partial_error_dev, BLOCKS * sizeof(double)));
    HANDLE_ERROR(hipMalloc((void**)&partial_mean_dev, BLOCKS * sizeof(double)));

    //calculating partial errors and means for the samples of XT
    euler_for_unknown<<<BLOCKS,THREADS>>>(partial_mean_dev, partial_error_dev);

    HANDLE_ERROR(hipMemcpy(partial_error, partial_error_dev, BLOCKS*sizeof(double), hipMemcpyDeviceToHost));
    HANDLE_ERROR(hipMemcpy(partial_mean, partial_mean_dev, BLOCKS*sizeof(double), hipMemcpyDeviceToHost));

    hipFree(partial_error_dev);
    hipFree(partial_mean_dev);

    HANDLE_ERROR(hipEventRecord(stop, 0));

    hipEventSynchronize(stop);
    float time;
    HANDLE_ERROR(hipEventElapsedTime(&time, start, stop));
    printf("GPU computations time: %3.1f ms\n", time);
    hipEventDestroy(start);
    hipEventDestroy(stop);

    //CPU usage for solving reduced problem of partial errors and means
    print_results(partial_mean, partial_error, BLOCKS);

    free(partial_mean);
    free(partial_error);

    //printing and additionally saving given number of samples from XT
    //useful for exploring the probability distribution
    get_solution_samples(10000);

    return 0;
}